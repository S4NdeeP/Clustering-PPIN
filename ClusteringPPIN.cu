#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include<math.h>
#include "convertDIPtoMat_CSR.cu"

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
/*
struct Matrix_CSR {
    int* columnIndices,*rowOffsets;
    float* values;
    int nonZeroValueCount;
    int rowCount;
};
*/

/**
 * CUDA kernel function that squares each element of the array.
 */
__global__ void square_row(struct Matrix_CSR* d_m,int inflationParameter) {
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<d_m->nonZeroValueCount) {
		d_m->values[i]=pow(d_m->values[i],inflationParameter);
		d_m->values[i]=((int)(d_m->values[i]*1000.0))/1000.0;

	}
}

__global__ void normalize_row(struct Matrix_CSR* d_m) {
	int i,j;
	float sum;
	i=blockIdx.x*blockDim.x+threadIdx.x;

	if(i<d_m->rowCount) {
		sum=0.0;

		for(j=d_m->rowOffsets[i];j<d_m->rowOffsets[i+1];j++) {
			sum=sum+d_m->values[j];
		}

		for(j=d_m->rowOffsets[i];j<d_m->rowOffsets[i+1];j++) {
			d_m->values[j]=d_m->values[j]/sum;
			d_m->values[i]=((int)(d_m->values[i]*1000.0))/1000.0;

		}
	}
}


Matrix_CSR * inflateRow(struct Matrix_CSR *d_m,int nonZeroValueCount,int rowCount,int inflationParameter) {
	square_row<<<((nonZeroValueCount+512)/512),512>>>(d_m,inflationParameter);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete

	normalize_row<<<((rowCount+512)/512),512>>>(d_m);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete

	return d_m;
}


__global__ void row_based_mult(int row,struct Matrix_CSR* primeValue,struct Matrix_CSR *subValue,float *tempColumn){
	int j,k,l;
	float tempSum=0.0;
	j=blockIdx.x*blockDim.x+threadIdx.x;
	if(j<primeValue->rowCount){

		for(k=primeValue->rowOffsets[row];k<primeValue->rowOffsets[row+1];k++) {
				for(l=subValue->rowOffsets[primeValue->columnIndices[k]];l<subValue->rowOffsets[primeValue->columnIndices[k]+1];l++  ) {
					if(j==subValue->columnIndices[l]) {
						tempSum=tempSum+primeValue->values[k]*subValue->values[l];
						break;
					}
					if(j<subValue->columnIndices[l]) {
						break;
					}
				}
		}
		tempColumn[j]=tempSum;
		}
	}

	__global__ void row_based_induction(int row,struct Matrix_CSR* d_m,float *tempValues,int *tempColumnIndices,int *tempRowOffsets,float *tempColumn,int *countResult){
	int i,count;
	tempRowOffsets[0]=0;
	count=tempRowOffsets[row];
		for(i=0;i<d_m->rowCount;i++){
			if(tempColumn[i]>=0.00001){
				tempValues[count]=tempColumn[i];
				tempValues[count]=((int)(tempValues[count]*1000.0))/1000.0;
				tempColumnIndices[count]=i;
				count=count+1;
			}
		}
		tempRowOffsets[row+1]=count;
	*countResult=count;
}

__global__ void copy_function(int row,float *newTempValues, int *newTempColumnIndices, float *tempValues, int *tempColumnIndices,int currentSize){
int i=blockIdx.x*blockDim.x+threadIdx.x;
	 if(i<currentSize){
		newTempValues[i]=tempValues[i];
		newTempColumnIndices[i]=tempColumnIndices[i];
		}
}

__global__ void reinitialize_function(struct Matrix_CSR* d_m,float *tempValues,int *tempColumnIndices,int *tempRowOffsets,int nonZeroValueCount){

	d_m->values=tempValues;
	d_m->columnIndices=tempColumnIndices;
	d_m->rowOffsets=tempRowOffsets;
	d_m->nonZeroValueCount=nonZeroValueCount;

}


int matrix_multiplication(struct Matrix_CSR *primeValue,struct Matrix_CSR *subValue,int rowCount,int clearFlag)
{
	int i,assignedSize,currentSize,startFree=0;
	float *tempColumn, *tempValues, *newTempValues, *flTempFree;
	int *tempColumnIndices, *newTempColumnIndices, *tempRowOffsets, *intTempFree1, *countResult;
	assignedSize=rowCount;

	struct Matrix_CSR *temp;
	temp=(struct Matrix_CSR*)malloc(sizeof(struct Matrix_CSR));


	CUDA_CHECK_RETURN(hipMalloc((void**) &tempColumn, sizeof(float)*rowCount));
	CUDA_CHECK_RETURN(hipMalloc((void**) &tempValues, sizeof(float)*rowCount));
	CUDA_CHECK_RETURN(hipMalloc((void**) &tempColumnIndices, sizeof(int)*rowCount));
	CUDA_CHECK_RETURN(hipMalloc((void**) &tempRowOffsets, sizeof(int)*(rowCount+1)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &countResult, sizeof(int)));


	for(i=0;i<rowCount;i++){
		hipMemset(tempColumn,0,sizeof(float)*rowCount);
		row_based_mult<<<((rowCount+512)/512),512>>>(i,primeValue,subValue,tempColumn);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
		row_based_induction<<<1,1>>>(i,primeValue,tempValues,tempColumnIndices,tempRowOffsets,tempColumn,countResult);
		CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
		CUDA_CHECK_RETURN(hipMemcpy(&currentSize,countResult,sizeof(int),hipMemcpyDeviceToHost));
		if((assignedSize-currentSize)<(rowCount)){
			assignedSize=2*assignedSize;
			CUDA_CHECK_RETURN(hipMalloc((void**) &newTempValues, sizeof(float)*assignedSize));
			CUDA_CHECK_RETURN(hipMalloc((void**) &newTempColumnIndices, sizeof(int)*assignedSize));
			copy_function<<<((currentSize+512)/512),512>>>(i,newTempValues,newTempColumnIndices,tempValues,tempColumnIndices,currentSize);
			CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
			flTempFree=tempValues;
			intTempFree1=tempColumnIndices;
			tempValues=newTempValues;
			tempColumnIndices=newTempColumnIndices;
			CUDA_CHECK_RETURN(hipFree((void*) flTempFree));
			CUDA_CHECK_RETURN(hipFree((void*) intTempFree1));
			}

	}

	CUDA_CHECK_RETURN(hipMemcpy(temp,primeValue,sizeof(struct Matrix_CSR), hipMemcpyDeviceToHost));

	reinitialize_function<<<1,1>>>(primeValue,tempValues,tempColumnIndices,tempRowOffsets,currentSize);

	if(clearFlag)
{
	CUDA_CHECK_RETURN(hipFree((void*) temp->columnIndices));
	CUDA_CHECK_RETURN(hipFree((void*) temp->rowOffsets));
	CUDA_CHECK_RETURN(hipFree((void*) temp->values));
	CUDA_CHECK_RETURN(hipFree((void*) tempColumn));
	CUDA_CHECK_RETURN(hipFree((void*) countResult));
}
	return currentSize;

}


int expand(struct Matrix_CSR *d_m,int rowCount,int expansionParameter) {
	struct Matrix_CSR *temp,*variable;
	int binary[8],i=0;
	int currentSize,fastExpQuotient,clearFlag=0;
	temp=(struct Matrix_CSR*)malloc(sizeof(struct Matrix_CSR));
	CUDA_CHECK_RETURN(hipMalloc((void**) &variable,sizeof(struct Matrix_CSR)));
	CUDA_CHECK_RETURN(hipMemcpy(temp,d_m,sizeof(struct Matrix_CSR), hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(variable,temp,sizeof(struct Matrix_CSR), hipMemcpyHostToDevice));

	fastExpQuotient=expansionParameter;

	while(fastExpQuotient)
	{
		binary[i++]=fastExpQuotient%2;
		fastExpQuotient=fastExpQuotient/2;
	}

	i=i-2;
	while(i>=0)
	{
		currentSize=matrix_multiplication(d_m,d_m,rowCount,clearFlag);
		clearFlag=1;
		if(binary[i]==1)
		{
			currentSize=matrix_multiplication(d_m,variable,rowCount,clearFlag);
		}
		i--;
	}

	return currentSize;
}



__global__ void converge(struct Matrix_CSR *d_m,int *convergeResult) {
	int i,j;
	float temp_val;

	i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<d_m->rowCount) {
		temp_val=d_m->values[d_m->rowOffsets[i]];
		for(j=d_m->rowOffsets[i];j<d_m->rowOffsets[i+1];j++) {
			if(temp_val!=d_m->values[j]) {
				convergeResult[i]=0;
				return;
			}
		}
		convergeResult[i]=1;
	}
}

Matrix_CSR * MCL(struct Matrix_CSR *m,int inflationParameter,int expansionOperator) {
	int noOfNonZeroValues, *converged;
	struct Matrix_CSR *d_m = NULL;
	struct Matrix_CSR *cluster=NULL;
	int* d_rowOffsets,*d_columnIndices, *convergeResult;
	float *d_values;

	cluster=(struct Matrix_CSR*)malloc(sizeof(struct Matrix_CSR));

	CUDA_CHECK_RETURN(hipMalloc((void**) &d_rowOffsets, sizeof(int)*(m->rowCount+1)));
	CUDA_CHECK_RETURN(hipMemcpy(d_rowOffsets, m->rowOffsets,sizeof(int)*(m->rowCount+1),hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipMalloc((void**) &d_columnIndices, sizeof(int)*m->nonZeroValueCount));
	CUDA_CHECK_RETURN(hipMemcpy(d_columnIndices, m->columnIndices,sizeof(int)*m->nonZeroValueCount,hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipMalloc((void**) &d_values, sizeof(float)*m->nonZeroValueCount));
	CUDA_CHECK_RETURN(hipMemcpy(d_values, m->values,sizeof(float)*m->nonZeroValueCount,hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipMalloc((void**) &convergeResult, sizeof(int)*m->rowCount));
	converged=(int*)malloc(sizeof(int)*m->rowCount); //????

	Matrix_CSR* temp=(struct Matrix_CSR*)malloc(sizeof(struct Matrix_CSR));
	temp->columnIndices=d_columnIndices;
	temp->rowOffsets=d_rowOffsets;
	temp->values=d_values;
	temp->rowCount=m->rowCount;
	temp->nonZeroValueCount=m->nonZeroValueCount;

	noOfNonZeroValues=m->nonZeroValueCount;
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_m,sizeof(struct Matrix_CSR)));
	CUDA_CHECK_RETURN(hipMemcpy(d_m,temp,sizeof(struct Matrix_CSR),hipMemcpyHostToDevice));
	normalize_row<<<((m->rowCount+512)/512),512>>>(d_m);

	int i,kill=0,flag=0;

	do {
		kill++;
		printf("Iteration:%d   Non zero value count:%d\n",kill,noOfNonZeroValues);
		noOfNonZeroValues=expand(d_m,m->rowCount,expansionOperator);
		inflateRow(d_m,noOfNonZeroValues,m->rowCount,inflationParameter);
		converge<<<((m->rowCount+512)/512),512>>>(d_m,convergeResult);
		CUDA_CHECK_RETURN(hipMemcpy(converged,convergeResult,sizeof(int)*m->rowCount,hipMemcpyDeviceToHost));
		for(i=0;i<m->rowCount;i++){
			if(converged[i]==0){
				flag=0;
				break;
			}
			flag=1;
		}
		if(kill>100) {
			printf("Didnt converge....try with other parameters...");
			break;
		}
} while(!flag);

	free(m->values);
	free(m->columnIndices);

	m->values=(float*)malloc(sizeof(float)*noOfNonZeroValues);
	m->columnIndices=(int*)malloc(sizeof(int)*noOfNonZeroValues);


	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipMemcpy(cluster,d_m,sizeof(struct Matrix_CSR), hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(m->values,cluster->values,sizeof(float)*noOfNonZeroValues, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(m->columnIndices,cluster->columnIndices,sizeof(int)*noOfNonZeroValues, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(m->rowOffsets,cluster->rowOffsets,sizeof(int)*(m->rowCount+1), hipMemcpyDeviceToHost));
	m->nonZeroValueCount=noOfNonZeroValues;

	CUDA_CHECK_RETURN(hipFree((void*) d_m));
	CUDA_CHECK_RETURN(hipFree((void*) cluster->columnIndices));
	CUDA_CHECK_RETURN(hipFree((void*) cluster->rowOffsets));
	CUDA_CHECK_RETURN(hipFree((void*) cluster->values));
	CUDA_CHECK_RETURN(hipFree((void*) convergeResult));
	return m;
}

void writeCSRToFile(struct Matrix_CSR* m) {
    FILE* fp=fopen("cluster.mcsr","w");
    printf("Cluster wrote to file...\n");
    int i;
    //printf("\n Non Zero Count %d",m->nonZeroValueCount);
    for(i=0;i<m->nonZeroValueCount;++i) {
        fprintf(fp,"%f ",m->values[i]);
    }

    fprintf(fp,"\n");
    for(i=0;i<m->nonZeroValueCount;++i) {
        fprintf(fp,"%d ",m->columnIndices[i]);
    }

    fprintf(fp,"\n");
    for(i=0;i<(m->rowCount)+1;++i) {
        fprintf(fp,"%d ",m->rowOffsets[i]);
    }
}

int convertToInt(char* string) {
	int i,value=0;
	for(i=0;string[i]!='\0';++i) {
		value=value*10+((int)string[i]-48);
	}
	return value;
}


/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */

/*
struct Matrix_CSR* readFromFile(char* file) {
	FILE* fp=fopen(file,"r");
	Matrix_CSR* m=(struct Matrix_CSR*)malloc(sizeof(struct Matrix_CSR));
	printf("\nIn read");
	fscanf(fp,"%d",&m->nonZeroValueCount);
	fscanf(fp,"%d",&m->rowCount);


	m->columnIndices=(int*)malloc(sizeof(int)*m->nonZeroValueCount);
	m->rowOffsets=(int*)malloc(sizeof(int)*(m->rowCount+1));
	m->values=(float*)malloc(sizeof(float)*m->nonZeroValueCount);
	int i;
	for(i=0;i<m->nonZeroValueCount;++i) {
		fscanf(fp,"%f",&m->values[i]);
	}

	for(i=0;i<m->nonZeroValueCount;++i) {
			fscanf(fp,"%d",&m->columnIndices[i]);
		}

	for(i=0;i<=m->rowCount;++i) {
			fscanf(fp,"%d",&m->rowOffsets[i]);
		}
	printf("\nOut read");
	return m;
}
*/

void writeClusterToFile(struct Matrix_CSR* cluster,char NodeIdTable[][20]) {

	struct bucket {
		int rowNumber;
		struct bucket *next;
	};

	struct bucket *array[cluster->rowCount], *temp;
	int i,j;

	for(i=0;i<cluster->rowCount;i++) {
		array[i]=NULL;
	}

	for(i=0;i<cluster->rowCount;i++)
	{
		for(j=cluster->rowOffsets[i];j<cluster->rowOffsets[i+1];j++)
		{
			if(array[cluster->columnIndices[j]]==NULL)
			{
				array[cluster->columnIndices[j]]=(struct bucket*)malloc(sizeof(struct bucket));
				array[cluster->columnIndices[j]]->rowNumber=i;
				array[cluster->columnIndices[j]]->next=NULL;
			}
			else
			{
				temp=(struct bucket*)malloc(sizeof(struct bucket));
				temp->rowNumber=i;
				temp->next=array[cluster->columnIndices[j]];
				array[cluster->columnIndices[j]]=temp;
			}

		}
	}

	for(i=0;i<cluster->rowCount;++i) {
		for(j=i+1;j<cluster->rowCount;++j) {
			struct bucket* node1,*node2;
			for(node1=array[i],node2=array[j];node1!=NULL&&node2!=NULL;node1=node1->next,node2=node2->next) {
				if(node1->rowNumber!=node2->rowNumber) {
					break;
				}
			}
			if(node1==NULL&&node2==NULL) {
				array[j]=NULL;
			}
		}
	}

	FILE* fp=fopen("ClusterSets","w");

	for(i=0;i<cluster->rowCount;++i) {
		struct bucket* node;
		if(array[i]==NULL) {
			continue;
		}
		for(node=array[i];node!=NULL;node=node->next) {
			fprintf(fp,"%s ",NodeIdTable[node->rowNumber]);
			//fprintf(fp,"%d ",node->rowNumber);
		}
		fprintf(fp,"\n");
	}
	printf("Finished Writing Clusters to File...\n");
}

int main(int argc, char** argv) {
	int inflationParameter,expansionOperator;
	struct Matrix_CSR *m,*cluster;
	char NodeIdTable[10000][20];
	m=convertDIPtoMat_CSR(argv[1],NodeIdTable);

	if(argc<4) {
		printf("Too few arguments");
		exit(0);
	}

	inflationParameter=convertToInt(argv[2]);
	expansionOperator=convertToInt(argv[3]);

	cluster=MCL(m,inflationParameter,expansionOperator);
	printf("Clusters created...");
	writeClusterToFile(cluster,NodeIdTable);
	//writeCSRToFile(cluster);
	return 0;
}
